#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <imageManager.h>
#include <imageUtils.cuh>

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

using DT = float;

TextureInfo createTextureObjectFrom2DArray(const ImageInfo<DT>& ii) {
    TextureInfo ti;

    // Size info
    ti.size = { ii.width, ii.height, 1 };

    //Texture Data settings
    ti.texChannelDesc = hipCreateChannelDesc<float>();  // hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors(hipMallocArray(&ti.texArrayData, &ti.texChannelDesc, ii.width, ii.height));
    checkCudaErrors(hipMemcpyToArray(ti.texArrayData, 0, 0, ii.dPtr, ii.pitch * ii.height, hipMemcpyDeviceToDevice));

    // Specify texture resource
    ti.resDesc.resType = hipResourceTypeArray;
    ti.resDesc.res.array.array = ti.texArrayData;

    // Specify texture object parameters
    ti.texDesc.addressMode[0] = hipAddressModeClamp;
    ti.texDesc.addressMode[1] = hipAddressModeClamp;
    ti.texDesc.filterMode = hipFilterModePoint;
    ti.texDesc.readMode = hipReadModeElementType;
    ti.texDesc.normalizedCoords = false;

    // Create texture object
    checkCudaErrors(hipCreateTextureObject(&ti.texObj, &ti.resDesc, &ti.texDesc, nullptr));

    return ti;
}

ImageInfo<uchar3> allocateNormalMap(ImageInfo<DT> hMap) {
    ImageInfo<uchar3> nMap {
            hMap.width,
            hMap.height,
            hMap.pitch,
            nullptr
    };
    hipMallocPitch((void**)&nMap.dPtr, &nMap.pitch, hMap.width * sizeof(uchar3), hMap.height);
    return nMap;
}

__device__ float    getSobelX(const hipTextureObject_t &tHMap, float x, float y) {
    float res = 0.0f;
    res -= tex2D<float>(tHMap, x-1, y-1);
    res -= 2 * tex2D<float>(tHMap, x-1, y);
    res -= tex2D<float>(tHMap, x-1, y+1);

    res += tex2D<float>(tHMap, x+1, y-1);
    res += 2 * tex2D<float>(tHMap, x+1, y);
    res += tex2D<float>(tHMap, x+1, y+1);

    return res;
}

__device__ float getSobelY(const hipTextureObject_t &tHMap, float x, float y) {
    float res = 0.0f;
    res -= tex2D<float>(tHMap, x-1, y+1);
    res -= 2 * tex2D<float>(tHMap, x, y+1);
    res -= tex2D<float>(tHMap, x+1, y+1);

    res += tex2D<float>(tHMap, x-1, y-1);
    res += 2 * tex2D<float>(tHMap, x, y-1);
    res += tex2D<float>(tHMap, x+1, y-1);

    return res;
}

__device__ float3 normalizeVec(float x, float y, float z) {
    float l = sqrt(x*x + y*y + z*z);
    return float3 {x/l, y/l, z/l};
}

__device__ uchar3 getNormal(const hipTextureObject_t &tHMap, unsigned int x, unsigned int y) {
    auto normalizedNorm = normalizeVec(
            getSobelX(tHMap, (float) x, (float) y),
            getSobelY(tHMap, (float) x, (float) y),
            5);

    //SHIFT TO UPPER RIGHT QUADRANT
    normalizedNorm.x = (normalizedNorm.x + 1.0f) / 2.0f;
    normalizedNorm.y = (normalizedNorm.y + 1.0f) / 2.0f;
    //normalizedNorm.z = (normalizedNorm.z + 1.0f) / 2.0f;

    // CONVERT FLOAT VECTOR INTO UCHAR;
    return uchar3 {
        (unsigned char) (normalizedNorm.z * 255.0f),
        (unsigned char) (normalizedNorm.y * 255.0f),
        (unsigned char) (normalizedNorm.x * 255.0f)
    };
}

__global__ void calculateSobel(const hipTextureObject_t tHMap, const ImageInfo<uchar3> nMap) {
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t tCount = gridDim.x * blockDim.x;
    uint32_t dataSize = nMap.width * nMap.height;

    while (tid < dataSize) {
        unsigned int x = tid % nMap.width;
        unsigned int y = tid / nMap.width;

        auto rowStart = (uchar3*) ((char*) nMap.dPtr + y * nMap.pitch);
        rowStart[x] = getNormal(tHMap, x, y);
        tid += tCount;
    }

}

void saveTexImage(const char* imageFileName, const uint32_t dstWidth, const uint32_t dstHeight, const uint32_t dstPitch, const uchar3* dstData) {
    FIBITMAP* tmp = FreeImage_Allocate(dstWidth, dstHeight, 24);
    unsigned int tmpPitch = FreeImage_GetPitch(tmp);					// FREEIMAGE align row data ... You have to use pitch instead of width
    checkCudaErrors(hipMemcpy2D(FreeImage_GetBits(tmp), tmpPitch, dstData, dstPitch, dstWidth * 3, dstHeight, hipMemcpyDeviceToHost));
    ImageManager::GenericWriter(tmp, imageFileName, FIF_BMP);
    FreeImage_Unload(tmp);
}


int main() {
    initializeCUDA(deviceProp);
    FreeImage_Initialise();

    ImageInfo<DT> hMap;
    ImageInfo<uchar3> nMap;
    prepareData<false>("/home/pavel/prg/cpp/ParallelApplications2/images/terrain3Kx3K.tif", hMap);
    nMap = allocateNormalMap(hMap);

    TextureInfo tiHMap = createTextureObjectFrom2DArray(hMap);

    dim3 block { 10, 1 ,1 };
    dim3 grid { 64, 1, 1 };

    calculateSobel<<<block, grid>>>(tiHMap.texObj, nMap);

    auto hNMap = static_cast<uchar3*>(::operator new (nMap.width * nMap.height * sizeof(uchar3)));
    error = hipMemcpy2D(hNMap, sizeof(uchar3) * nMap.width, nMap.dPtr, nMap.pitch, nMap.width, nMap.height, hipMemcpyDeviceToHost);

   // saveTexImage("/home/pavel/res.tif", nMap.width, nMap.height, sizeof(uchar3) * nMap.width, hNMap);
    saveTexImage("/home/pavel/res.tif", nMap.width, nMap.height, nMap.pitch, nMap.dPtr);

    /*
    int *m = static_cast<int*>(::operator new (sizeof(int)*rows*cols));;
    hipMemcpy2D(m, sizeof(int) * cols, dM, dPitch, cols * sizeof(int), rows, hipMemcpyDeviceToHost);
    */
    if (tiHMap.texObj) checkCudaErrors(hipDestroyTextureObject(tiHMap.texObj));
    if (tiHMap.texArrayData) checkCudaErrors(hipFreeArray(tiHMap.texArrayData));

    FreeImage_DeInitialise();
    if (hMap.dPtr) hipFree(hMap.dPtr);
    if (nMap.dPtr) hipFree(nMap.dPtr);
    return 0;
}
