#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

#include <fstream>
#include <regex>
#include <string>
#include <algorithm>
#include <random>
#include <chrono>

#include <cudaDefs.h>
#include <FreeImage.h>
#include <imageManager.h>

using namespace std;

//TODO -> update TPB_1D and TPB_2D values
constexpr unsigned int TPB_1D = 8;									// ThreadsPerBlock in one dimension
constexpr unsigned int TPB_2D =  TPB_1D * TPB_1D;					// ThreadsPerBlock = TPB_1D*TPB_1D (2D block)

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

std::mt19937 generator(std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count());

struct GLData {
    unsigned int imageWidth;
    unsigned int imageHeight;
    unsigned int imageBPP;
    unsigned int imagePitch;

    unsigned int pboID;
    unsigned int textureID;
    unsigned int viewportWidth = 1024;
    unsigned int viewportHeight = 1024;
};

struct CudaData {
    hipTextureDesc			texDesc;				// Texture descriptor used to describe texture parameters

    hipArray_t				texArrayData;			// Source texture data
    hipResourceDesc		resDesc;				// A resource descriptor for obtaining the texture data
    hipChannelFormatDesc	texChannelDesc;			// Texture channel descriptor to define channel bytes
    hipTextureObject_t		texObj;					// Cuda Texture Object to be produces

    hipGraphicsResource_t  texResource;
    hipGraphicsResource_t	pboResource;

    CudaData() {
        memset(this, 0, sizeof(CudaData));			// DO NOT DELETE THIS !!!
    }
};

struct Settings {
    int leaders;
    int followers;
    string heightMap;
    int heightmapGridX;
    int heightmapGridY;
    float leaderRadius;
    float speedFactor;
    string outputFile;

    string getAtribVal(string &str, const string& atribName) {
        smatch m;
        regex rt(atribName + "\":([\\S\\s]+?(?=,|}))");
        regex_search(str, m, rt);
        string val = m.str().substr(atribName.size()+2);
        if (val.at(0) == '\"') {
            val = val.substr(1, val.size()-2);
        }
        return val;
    }

    Settings() = default;

    Settings(const string& path) {
        fstream inStream(path);
        std::string str((std::istreambuf_iterator<char>(inStream)),std::istreambuf_iterator<char>());
        str.erase(remove_if(str.begin(), str.end(), ::isspace), str.end());

        this->leaders = std::stoi(getAtribVal(str, "leaders"));
        this->followers = std::stoi(getAtribVal(str, "followers"));
        this->heightmapGridX = std::stoi(getAtribVal(str, "heightmapGridX"));
        this->heightmapGridY = std::stoi(getAtribVal(str, "heightmapGridY"));
        this->leaderRadius = std::stof(getAtribVal(str, "leaderRadius"));
        this->speedFactor = std::stof(getAtribVal(str, "speedFactor"));
        this->heightMap = getAtribVal(str, "heightmap");
        this->outputFile = getAtribVal(str, "outputFile");
    }
};
Settings settings;

struct HeightMap {
    GLData glData;
    CudaData cudaData;

    void prepareGlObjects(const char* imageFileName) {  //alokovani zdroju na karte
        FIBITMAP* tmp = ImageManager::GenericLoader(imageFileName, 0);
        glData.imageWidth = FreeImage_GetWidth(tmp);
        glData.imageHeight = FreeImage_GetHeight(tmp);
        glData.imageBPP = FreeImage_GetBPP(tmp);
        glData.imagePitch = FreeImage_GetPitch(tmp);

        //OpenGL Texture
        glEnable(GL_TEXTURE_2D);
        glGenTextures(1, &glData.textureID);
        glBindTexture(GL_TEXTURE_2D, glData.textureID);

        //WARNING: Just some of inner format are supported by CUDA!!!
        glTexImage2D(GL_TEXTURE_2D, 0, GL_LUMINANCE, glData.imageWidth, glData.imageHeight, 0, GL_RED, GL_UNSIGNED_BYTE, FreeImage_GetBits(tmp));
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);


        glBindTexture(GL_TEXTURE_2D, 0);
        FreeImage_Unload(tmp);

        glGenBuffers(1, &glData.pboID);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, glData.pboID);														// Make this the current UNPACK buffer (OpenGL is state-based)
        glBufferData(GL_PIXEL_UNPACK_BUFFER, glData.imageWidth * glData.imageHeight * 4, NULL, GL_DYNAMIC_COPY);	// Allocate data for the buffer. 4-channel 8-bit image
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
    }

    void initCUDAObjects() {
        // Register Image to cuda tex resource
        checkCudaErrors(hipGraphicsGLRegisterImage(
                &cudaData.texResource,
                glData.textureID,
                GL_TEXTURE_2D,
                hipGraphicsRegisterFlags::hipGraphicsRegisterFlagsReadOnly
        ));

        // Map reousrce and retrieve pointer to undelying array data
        checkCudaErrors(hipGraphicsMapResources(1, &cudaData.texResource, 0)); //OPENGL, pls nepracuj ted s tou texturou
        checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&cudaData.texArrayData, cudaData.texResource, 0, 0));    //z resourcu chci tahat pixelova data

        // Set resource descriptor
        cudaData.resDesc.resType = hipResourceType::hipResourceTypeArray;
        cudaData.resDesc.res.array.array = cudaData.texArrayData;

        // Set Texture Descriptor: Tex Units will know how to read the texture
        cudaData.texDesc.readMode = hipReadModeElementType;
        cudaData.texDesc.normalizedCoords = false;
        cudaData.texDesc.filterMode = hipFilterModePoint;
        cudaData.texDesc.addressMode[0] = hipAddressModeClamp;
        cudaData.texDesc.addressMode[1] = hipAddressModeClamp;

        // Set Channel Descriptor: How to interpret individual bytes
        checkCudaErrors(hipGetChannelDesc(&cudaData.texChannelDesc, cudaData.texArrayData));

        // Create CUDA Texture Object
        checkCudaErrors(hipCreateTextureObject(&cudaData.texObj, &cudaData.resDesc, &cudaData.texDesc, nullptr));

        // Unmap resource: Release the resource for OpenGL
        checkCudaErrors(hipGraphicsUnmapResources(1, &cudaData.texResource, 0));

        // Register PBO
        checkCudaErrors(hipGraphicsGLRegisterBuffer(
                &cudaData.pboResource,
                glData.pboID,
                hipGraphicsRegisterFlags::hipGraphicsRegisterFlagsWriteDiscard
        ));
    }

    void init(const string& path) {
        prepareGlObjects(path.c_str());
        initCUDAObjects();
    }

    ~HeightMap() {
        checkCudaErrors(hipGraphicsUnregisterResource(this->cudaData.texResource));
        checkCudaErrors(hipGraphicsUnregisterResource(this->cudaData.pboResource));

        if (this->glData.textureID > 0)
            glDeleteTextures(1, &this->glData.textureID);
        if (this->glData.pboID > 0)
            glDeleteBuffers(1, &this->glData.pboID);
    }
};
HeightMap hMap;
unsigned int overlayTexId;

struct Particle {
    float x, y;         // POSITION
    float v_x { 0.0f }; // VELOCITY IN DIRECTION X
    float v_y { 0.0f }; // VELOCITY IN DIRECTION Y
};
Particle* dLeaders;
Particle* dFollowers;

std::vector<Particle> generateParticles(int n) {
    std::vector<Particle> result;

    std::uniform_real_distribution<float> dis(0.0, 1.0);
    for (int i = 0; i < n; i++) {
        result.push_back(Particle{
            dis(generator) * hMap.glData.imageWidth,
            dis(generator) * hMap.glData.imageHeight,
            0.0f, 0.0f
        });
    }

    return result;
}

#pragma region --- CUDA ---

__global__ void clearPBO(unsigned char* pbo, const unsigned int pboWidth, const unsigned int pboHeight) {
    unsigned int tx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx >= pboWidth || ty > pboHeight) return;
    unsigned int pboIdx = ((ty * pboWidth) + tx) * 4;

    pbo[pboIdx++] = 0;
    pbo[pboIdx++] = 0;
    pbo[pboIdx++] = 0;
    pbo[pboIdx]   = 0;
}

__global__ void renderParticles(uchar3 color, Particle* particles, int particleCount, unsigned char* pbo, const unsigned int pboWidth, const unsigned int pboHeight) {
    unsigned int tx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int jump = blockDim.x * gridDim.x;

    while (tx < particleCount) {
        Particle p = particles[tx];
        unsigned int pboIdx = ((floor(p.y) * pboWidth) + floor(p.x)) * 4;
        pbo[pboIdx++] = color.x;
        pbo[pboIdx++] = color.y;
        pbo[pboIdx++] = color.z;
        pbo[pboIdx]   = 255;

        tx += jump;
    }
}

void cudaWorker() {
    // Map GL resources
    checkCudaErrors(hipGraphicsMapResources(1, &hMap.cudaData.texResource, 0));
    checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&hMap.cudaData.texArrayData, hMap.cudaData.texResource, 0, 0));

    // TODO -> move pbo resource to be part of overlay texture
    checkCudaErrors(hipGraphicsMapResources(1, &hMap.cudaData.pboResource, 0));
    unsigned char* pboData;
    size_t pboSize;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&pboData, &pboSize, hMap.cudaData.pboResource));

    {   // CLEAR PBO
        dim3 block(TPB_1D, TPB_1D, 1);
        dim3 grid((hMap.glData.imageWidth + TPB_1D - 1) / TPB_1D, (hMap.glData.imageHeight + TPB_1D - 1) / TPB_1D, 1);
        clearPBO<<<grid, block>>>(pboData, hMap.glData.imageWidth, hMap.glData.imageHeight);
    };

    {   // PUT PARTCLES INTO PBO
        constexpr uchar3 leaderColor = {255, 0, 0};
        constexpr uchar3 followerColor = {0, 0, 255};

        //TODO -> adjust block and grid sizes
        dim3 block(128, 1, 1);
        dim3 grid(1, 1, 1);
        renderParticles<<<grid, block>>>(leaderColor, dLeaders, settings.leaders, pboData, hMap.glData.imageWidth, hMap.glData.imageHeight);
        renderParticles<<<grid, block>>>(followerColor, dFollowers, settings.followers, pboData, hMap.glData.imageWidth, hMap.glData.imageHeight);
    };

    // TODO -> Run kernel


    // Unmap GL Resources
    checkCudaErrors(hipGraphicsUnmapResources(1, &hMap.cudaData.texResource, 0));
    checkCudaErrors(hipGraphicsUnmapResources(1, &hMap.cudaData.pboResource, 0));

    // This updates GL texture from PBO
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, hMap.glData.pboID);
    glBindTexture(GL_TEXTURE_2D, overlayTexId);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, hMap.glData.imageWidth, hMap.glData.imageHeight, GL_RGBA, GL_UNSIGNED_BYTE, NULL);   //Source parameter is NULL, Data is coming from a PBO, not host memory
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
}

#pragma endregion

#pragma region --- OPEN_GL ---

void my_display() {
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    glEnable(GL_TEXTURE_2D);

    //glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, hMap.glData.textureID);
    glBegin(GL_QUADS);
    glTexCoord2d(0, 0);		glVertex2d(0, 0);
    glTexCoord2d(1, 0);		glVertex2d( hMap.glData.viewportWidth, 0);
    glTexCoord2d(1, 1);		glVertex2d( hMap.glData.viewportWidth,  hMap.glData.viewportHeight);
    glTexCoord2d(0, 1);		glVertex2d(0,  hMap.glData.viewportHeight);
    glEnd();

    //glActiveTexture(GL_TEXTURE1);
    glBindTexture(GL_TEXTURE_2D, overlayTexId);
    glBegin(GL_QUADS);
    glTexCoord2d(0, 0);		glVertex2d(0, 0);
    glTexCoord2d(1, 0);		glVertex2d( hMap.glData.viewportWidth, 0);
    glTexCoord2d(1, 1);		glVertex2d( hMap.glData.viewportWidth,  hMap.glData.viewportHeight);
    glTexCoord2d(0, 1);		glVertex2d(0,  hMap.glData.viewportHeight);
    glEnd();


    glDisable(GL_TEXTURE_2D);

    glFlush();
    glutSwapBuffers();
}

void my_resize(GLsizei w, GLsizei h) {
    hMap.glData.viewportWidth = w;
    hMap.glData.viewportHeight = h;

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    glViewport(0, 0, hMap.glData.viewportWidth, hMap.glData.viewportHeight);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, hMap.glData.viewportWidth, 0, hMap.glData.viewportHeight);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glutPostRedisplay();
}

void my_idle() {
    cudaWorker();
    glutPostRedisplay();
}

void initGL(int argc, char** argv) {
    glutInit(&argc, argv);

    glutInitDisplayMode(GLUT_RGBA | GLUT_DEPTH | GLUT_DOUBLE);
    glutInitWindowSize(hMap.glData.viewportWidth, hMap.glData.viewportHeight);
    glutInitWindowPosition(0, 0);
    glutSetOption(GLUT_RENDERING_CONTEXT, false ? GLUT_USE_CURRENT_CONTEXT : GLUT_CREATE_NEW_CONTEXT);
    glutCreateWindow(0);

    char m_windowsTitle[512];
    snprintf(m_windowsTitle, 512, "SimpleView | context %s | renderer %s | vendor %s ",
             (const char*)glGetString(GL_VERSION),
             (const char*)glGetString(GL_RENDERER),
             (const char*)glGetString(GL_VENDOR));
    glutSetWindowTitle(m_windowsTitle);

    glutDisplayFunc(my_display);
    glutReshapeFunc(my_resize);
    glutIdleFunc(my_idle);
    glutSetCursor(GLUT_CURSOR_CROSSHAIR);


    // initialize necessary OpenGL extensions
    glewInit();

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glShadeModel(GL_SMOOTH);
    glViewport(0, 0, hMap.glData.viewportWidth, hMap.glData.viewportHeight);

    glFlush();


}

#pragma endregion

void initOverlayTex() {
    //OpenGL Texture
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &overlayTexId);
    glBindTexture(GL_TEXTURE_2D, overlayTexId);

    std::vector<GLubyte> emptyData(hMap.glData.imageWidth * hMap.glData.imageHeight * 4, 128);
    //WARNING: Just some of inner format are supported by CUDA!!!
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, hMap.glData.imageWidth, hMap.glData.imageHeight, 0, GL_BGRA, GL_UNSIGNED_BYTE, &emptyData[0]);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glBindTexture(GL_TEXTURE_2D, 0);
}

int main(int argc, char* argv[]) {

    #pragma region initialize
    initializeCUDA(deviceProp);
    if (argc < 2) {
        printf("Please specify path to the configuration path");
        return 1;
    }
    settings = Settings(argv[1]);
    initGL(1, argv);

    // INITIALIZE HEIHGHT MAP
    hMap.init(settings.heightMap);
    auto a = glGetError();
    initOverlayTex();
    a = glGetError();



    // CREATE LEADERS AND COPY TO DEVICE
    auto leaders = generateParticles(settings.leaders);
    hipMalloc((void**)&dLeaders, settings.leaders * sizeof(Particle));
    hipMemcpy(dLeaders, leaders.data(), settings.leaders * sizeof(Particle), hipMemcpyHostToDevice);

    // CREATE FOLLOWERS AND COPY TO DEVICE
    auto followers = generateParticles(settings.followers);
    hipMalloc((void**)&dFollowers, settings.followers * sizeof(Particle));
    hipMemcpy(dFollowers, followers.data(), settings.followers * sizeof(Particle), hipMemcpyHostToDevice);

    #pragma endregion

    glutMainLoop();

    #pragma region clean_up
    if (dLeaders) hipFree(dLeaders);
    if (dFollowers) hipFree(dFollowers);
    #pragma endregion

    return 0;
}