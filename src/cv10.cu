#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <limits>
#include <benchmark.h>

#define __PRINT__  cout <<  __PRETTY_FUNCTION__ <<  endl

constexpr unsigned int TPB = 512;
constexpr unsigned int NO_BLOCKS = 46;
constexpr unsigned int N = 1 << 20;

constexpr int numberOfPasses = 1;

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

int *a, *b;
int *da, *db, *dGlobalMax;

__host__ void fillData(int *data, const int length) {
	for (int i=0; i<length; i++) {
		data[i]= i;
	}
	data[static_cast<int>(length * 0.5)] = length;
}

__host__ void fillData(int *data, const int length, const unsigned int value) {
	for (int i = 0; i<length; i++) {
		data[i] = i;
	}
}

__host__ void prepareData() {
	// paged-locked allocation
	constexpr unsigned int aSize = N * sizeof(int);
	constexpr unsigned int bSize = NO_BLOCKS * sizeof(int);

	hipHostAlloc((void**)&a, aSize, hipHostMallocDefault);
	hipHostAlloc((void**)&b, bSize, hipHostMallocDefault);

	fillData(a, N);
	fillData(b, NO_BLOCKS, INT_MIN);

	hipMalloc((void**)&da, aSize);
	hipMalloc((void**)&db, aSize);
	hipMalloc((void**)&dGlobalMax, sizeof(int));
	
	hipMemcpy(da, a, aSize, hipMemcpyHostToDevice);
	hipMemcpy(db, b, bSize, hipMemcpyHostToDevice);
}

__host__ void releaseData() {
	hipFree(da);
	hipFree(db);
	hipFree(dGlobalMax);

	hipHostFree(a);
	hipHostFree(b);
}

template<bool MAKE_IF>
__global__ void kernel0(const int* __restrict__ data, const unsigned int dataLength, int* __restrict__ globalMax) {
    // TODO: get max using atomic instruction

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int jump = gridDim.x * blockDim.x;

   const int* threadData = (int*)data + idx;

    while (idx < dataLength) {
        if constexpr(MAKE_IF) {
            if (*globalMax < *threadData) {
                atomicMax(globalMax, data[idx]);
            }
        } else {
            atomicMax(globalMax, data[idx]);
        }

        threadData += jump;
        idx += jump;
    }
}

template<bool MAKE_IF>
__global__ void kernel1(const int* __restrict__ data, const unsigned int dataLength, int* __restrict__ globalMax) {
    // TODO: get max using atomic instruction
    __shared__ int blockMax;

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.x == 0) blockMax = INT32_MIN;
    __syncthreads();

    const unsigned int jump = gridDim.x * blockDim.x;
    const int* threadData = data + idx;

    while (idx < dataLength) {
        if constexpr(MAKE_IF) {
            if (blockMax < *threadData) {
                atomicMax(&blockMax, data[idx]);
            }
        } else {
            atomicMax(&blockMax, data[idx]);
        }

        threadData += jump;
        idx += jump;
    }

    if (threadIdx.x == 0) atomicMax(globalMax, blockMax);
}

template<bool MAKE_IF>
__host__ void testKernel0() {
	dim3 blockSize(TPB, 1, 1);
	dim3 gridSize(getNumberOfParts(N, TPB), 1, 1);

	int globalMax = INT_MIN;

	auto test = [&]() {
		hipMemcpy(dGlobalMax, &globalMax, sizeof(int), hipMemcpyHostToDevice);
		kernel0<MAKE_IF><<<gridSize, blockSize>>> (da, N, dGlobalMax);
	};

	float gpuTime = GPUTIME(numberOfPasses, test());
    hipDeviceSynchronize();
	printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__ , gpuTime);

	hipMemcpy(&globalMax, dGlobalMax, sizeof(int), hipMemcpyDeviceToHost);
	printf("\nMaximum: %d\n", globalMax);
}


template<bool MAKE_IF>
__host__ void testKernel1() {
    dim3 blockSize(TPB, 1, 1);
    dim3 gridSize(getNumberOfParts(N, TPB), 1, 1);

    int globalMax = INT_MIN;

    auto test = [&]() {
        hipMemcpy(dGlobalMax, &globalMax, sizeof(int), hipMemcpyHostToDevice);
        kernel1<MAKE_IF><<<gridSize, blockSize>>> (da, N, dGlobalMax);
    };

    float gpuTime = GPUTIME(numberOfPasses, test());
    hipDeviceSynchronize();
    printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__ , gpuTime);

    hipMemcpy(&globalMax, dGlobalMax, sizeof(int), hipMemcpyDeviceToHost);
    printf("\nMaximum: %d\n", globalMax);
}



int main(int argc, char *argv[]) {
	initializeCUDA(deviceProp);
	prepareData();

	// TODO: CALL kernel 0
    testKernel0<true>();
    testKernel0<false>();
    testKernel1<true>();
    testKernel1<false>();

	releaseData();
	
	releaseData();
	return 0;
}
