#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <time.h>
#include <math.h>
#include <benchmark.h>

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

constexpr unsigned int N = 1 << 15;
constexpr unsigned int MEMSIZE = N * sizeof(unsigned int);
constexpr unsigned int NO_LOOPS = 100;
constexpr unsigned int TPB = 256;
constexpr unsigned int GRID_SIZE = (N + TPB - 1) / TPB;

constexpr unsigned int NO_TEST_PHASES = 10;

void fillData(unsigned int *data, const unsigned int length) {
	for (unsigned int i=0; i<length; i++) {
		data[i]= 1;
	}
}

void printData(const unsigned int *data, const unsigned int length) {
	if (data ==0) return;
	for (unsigned int i=0; i<length; i++) {
		printf("%u ", data[i]);
	}
}


__global__ void kernel(const unsigned int *a, const unsigned int *b, const unsigned int length, unsigned int *c) {
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int jump = gridDim.x * blockDim.x;

	//TODO:  thread block loop
	while (tid < length) {
		c[tid] = a[tid] + b[tid];
		tid += jump;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 1. - single stream, async calling </summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test1() {
	unsigned int *a, *b, *c;
	unsigned int *da, *db, *dc;

	// paged-locked allocation
	hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE,hipHostMallocDefault);

	fillData(a, NO_LOOPS * N);
	fillData(b, NO_LOOPS * N);

	// Data chunks on GPU
	hipMalloc( (void**)&da, MEMSIZE );
	hipMalloc( (void**)&db, MEMSIZE );
	hipMalloc( (void**)&dc, MEMSIZE );

	//TODO: create stream
	hipStream_t stream;
    hipStreamCreate(&stream);

	auto lambda = [&]() {
		unsigned int dataOffset = 0;
		for (int i = 0; i < NO_LOOPS; i++) {
            //TODO:  copy a->da, b->db
		    hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream);

			//TODO:  run the kernel in the stream
            dim3 grid(GRID_SIZE, 1, 1);
            dim3 block(TPB, 1, 1);
            kernel<<<grid, block, 0, stream>>>(da, db, N, dc);

			//TODO:  copy dc->c
            hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyKind::hipMemcpyDeviceToHost, stream);

			dataOffset += N;
		}
	};
	float gpuTime = GPUTIME(NO_TEST_PHASES, lambda());

	hipStreamSynchronize(stream); // wait for stream to finish
	hipStreamDestroy(stream);
	hipDeviceSynchronize();
	printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__, gpuTime);

	//printData(c, 100);
	
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 2. - two streams - depth first approach </summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test2() {
    unsigned int *a, *b, *c;
    unsigned int *da0, *db0, *dc0;
    unsigned int *da1, *db1, *dc1;

    // paged-locked allocation
    hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
    hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
    hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE,hipHostMallocDefault);

    fillData(a, NO_LOOPS * N);
    fillData(b, NO_LOOPS * N);

    // Data chunks on GPU
    hipMalloc( (void**)&da0, MEMSIZE );
    hipMalloc( (void**)&db0, MEMSIZE );
    hipMalloc( (void**)&dc0, MEMSIZE );

    hipMalloc( (void**)&da1, MEMSIZE );
    hipMalloc( (void**)&db1, MEMSIZE );
    hipMalloc( (void**)&dc1, MEMSIZE );

    //create stream
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    auto lambda = [&]() {
        unsigned int dataOffset = 0;
        int stream_id = 0;
        for (int i = 0; i < NO_LOOPS; i++) {
            //TODO:  copy a->da, b->db
            auto da = stream_id == 0 ? da0 : da1;
            auto db = stream_id == 0 ? db0 : db1;
            auto dc = stream_id == 0 ? dc0 : dc1;
            auto stream = stream_id == 0 ? stream0 : stream1;

            hipMemcpyAsync(da, &a[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(db, &b[dataOffset], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream);

            //TODO:  run the kernel in the stream
            dim3 grid(GRID_SIZE, 1, 1);
            dim3 block(TPB, 1, 1);
            kernel<<<grid, block, 0, stream>>>(da, db, N, dc);

            //TODO:  copy dc->c
            hipMemcpyAsync(&c[dataOffset], dc, MEMSIZE, hipMemcpyKind::hipMemcpyDeviceToHost, stream);

            dataOffset += N;
            stream_id = stream_id == 0 ? 1 : 0;
        }
    };
    float gpuTime = GPUTIME(NO_TEST_PHASES, lambda());

    hipStreamSynchronize(stream0); // wait for stream to finish
    hipStreamSynchronize(stream1); // wait for stream to finish
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipDeviceSynchronize();
    printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__, gpuTime);

    //printData(c, 100);

    hipFree(da0);
    hipFree(db0);
    hipFree(dc0);

    hipFree(da1);
    hipFree(db1);
    hipFree(dc1);

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Tests 3. - two streams - breadth first approach</summary>
////////////////////////////////////////////////////////////////////////////////////////////////////
void test3() {
    unsigned int *a, *b, *c;
    unsigned int *da0, *db0, *dc0;
    unsigned int *da1, *db1, *dc1;

    // paged-locked allocation
    hipHostAlloc((void**)&a, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
    hipHostAlloc((void**)&b, NO_LOOPS * MEMSIZE,hipHostMallocDefault);
    hipHostAlloc((void**)&c, NO_LOOPS * MEMSIZE,hipHostMallocDefault);

    fillData(a, NO_LOOPS * N);
    fillData(b, NO_LOOPS * N);

    // Data chunks on GPU
    hipMalloc( (void**)&da0, MEMSIZE );
    hipMalloc( (void**)&db0, MEMSIZE );
    hipMalloc( (void**)&dc0, MEMSIZE );

    hipMalloc( (void**)&da1, MEMSIZE );
    hipMalloc( (void**)&db1, MEMSIZE );
    hipMalloc( (void**)&dc1, MEMSIZE );

    //create stream
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    auto lambda = [&]() {
        unsigned int dataOffset0 = 0;
        unsigned int dataOffset1 = N;

        for (int i = 0; i < NO_LOOPS; i += 2) {
            hipMemcpyAsync(da0, &a[dataOffset0], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream0);
            hipMemcpyAsync(da1, &a[dataOffset1], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream1);

            hipMemcpyAsync(db0, &b[dataOffset0], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream0);
            hipMemcpyAsync(db1, &b[dataOffset1], MEMSIZE, hipMemcpyKind::hipMemcpyHostToDevice, stream1);

            //TODO:  run the kernel in the stream
            dim3 grid(GRID_SIZE, 1, 1);
            dim3 block(TPB, 1, 1);
            kernel<<<grid, block, 0, stream0>>>(da0, db0, N, dc0);
            kernel<<<grid, block, 0, stream1>>>(da1, db1, N, dc1);

            //TODO:  copy dc->c
            hipMemcpyAsync(&c[dataOffset0], dc0, MEMSIZE, hipMemcpyKind::hipMemcpyDeviceToHost, stream0);
            hipMemcpyAsync(&c[dataOffset1], dc1, MEMSIZE, hipMemcpyKind::hipMemcpyDeviceToHost, stream1);

            dataOffset0 += 2*N;
            dataOffset1 += 2*N;
        }
    };
    float gpuTime = GPUTIME(NO_TEST_PHASES, lambda());

    hipStreamSynchronize(stream0); // wait for stream to finish
    hipStreamSynchronize(stream1); // wait for stream to finish
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipDeviceSynchronize();
    printf("\x1B[93m[GPU time] %s: %f ms\033[0m\n", __PRETTY_FUNCTION__, gpuTime);

    //printData(c, 100);

    hipFree(da0);
    hipFree(db0);
    hipFree(dc0);

    hipFree(da1);
    hipFree(db1);
    hipFree(dc1);

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
}


int main(int argc, char *argv[]) {
	initializeCUDA(deviceProp);

	test1();
	test2();
	test3();

	return 0;
}
