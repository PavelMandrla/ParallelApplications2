#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <ctime>
#include <cmath>
#include <random>

//WARNING!!! Do not change TPB and NO_FORCES for this demo !!!
constexpr unsigned int TPB = 128;
constexpr unsigned int NO_FORCES = 256;     //TODO -> BONUS - zvednout pocet sil, ktere budu redukovat, na jakekoliv N
constexpr unsigned int NO_RAIN_DROPS = 1 << 20;

constexpr unsigned int MEM_BLOCKS_PER_THREAD_BLOCK = 8;

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

using namespace std;

float3 *createData(const unsigned int length) {
	random_device rd;
	mt19937_64 mt(rd());
	uniform_real_distribution<float> dist(0.0f, 1.0f);

    auto *data = static_cast<float3*>(::operator new(length * sizeof(float3)));
    float3* ptr = data;
    for (unsigned int i = 0; i < length; i++, ptr++) {
        *ptr = make_float3(dist(mt), dist(mt), dist(mt));
        //*ptr = make_float3(1,1,1);
    }
	return data;
}

void printData(const float3 *data, const unsigned int length) {
	if (data == 0) return;
	const float3 *ptr = data;
	for (unsigned int i = 0; i<length; i++, ptr++)
	{
		printf("%5.2f %5.2f %5.2f ", ptr->x, ptr->y, ptr->z);
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Sums the forces to get the final one using parallel reduction. 
/// 		    WARNING!!! The method was written to meet input requirements of our example, i.e. 128 threads and 256 forces  </summary>
/// <param name="dForces">	  	The forces. </param>
/// <param name="noForces">   	The number of forces. </param>
/// <param name="dFinalForce">	[in,out] If non-null, the final force. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void reduce(const float3 * __restrict__ dForces, const unsigned int noForces, float3* __restrict__ dFinalForce) {
	__shared__ float3 sForces[TPB];					//SEE THE WARNING MESSAGE !!!
	unsigned int tid = threadIdx.x;
	unsigned int next = TPB;						//SEE THE WARNING MESSAGE !!!

	float3* src = &sForces[tid];
	float3* src2 = (float3*)&dForces[tid + next];

	*src = dForces[tid];
	src->x += src2->x;
	src->y += src2->y;
	src->z += src2->z;

    __syncthreads();
    next >>= 1;                 //64
    if (tid >= next) return;
    src2 = src + next;
    src->x += src2->x;
    src->y += src2->y;
    src->z += src2->z;

    __syncthreads();
    while (next > 1) {      //32 AND DOWN
        next >>= 1;
        if (tid >= next) return;
        volatile float3* vsrc = &sForces[tid];
        volatile float3* vsrc2 = vsrc + next;
        vsrc->x += vsrc2->x;
        vsrc->y += vsrc2->y;
        vsrc->z += vsrc2->z;
    }

    if (tid == 0)
        *dFinalForce = sForces[0];

}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Adds the FinalForce to every Rain drops position. </summary>
/// <param name="dFinalForce">	The final force. </param>
/// <param name="noRainDrops">	The number of rain drops. </param>
/// <param name="dRainDrops"> 	[in,out] If non-null, the rain drops positions. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void add(const float3* __restrict__ dFinalForce, const unsigned int noRainDrops, float3* __restrict__ dRainDrops) {
	//TODO: Add the FinalForce to every Rain drops position.
}

int main(int argc, char *argv[]) {
	initializeCUDA(deviceProp);

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;

	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	hipEventRecord(startEvent, 0);

	float3 *hForces = createData(NO_FORCES);
	float3 *hDrops = createData(NO_RAIN_DROPS);

	float3 *dForces = nullptr;
	float3 *dDrops = nullptr;
	float3 *dFinalForce = nullptr;

	error = hipMalloc((void**)&dForces, NO_FORCES * sizeof(float3));
	error = hipMemcpy(dForces, hForces, NO_FORCES * sizeof(float3), hipMemcpyHostToDevice);

	error = hipMalloc((void**)&dDrops, NO_RAIN_DROPS * sizeof(float3));
	error = hipMemcpy(dDrops, hDrops, NO_RAIN_DROPS * sizeof(float3), hipMemcpyHostToDevice);

	error = hipMalloc((void**)&dFinalForce, sizeof(float3));

	KernelSetting ksReduce;
	ksReduce.dimBlock = dim3(TPB, 1, 1);
	//TODO: ... Set ksReduce

	KernelSetting ksAdd;
	//TODO: ... Set ksAdd

	//check sum
	float3 checkSum = make_float3(0,0,0);
	for (int i = 0; i < NO_FORCES; i++) {
	    checkSum.x += hForces[i].x;
	    checkSum.y += hForces[i].y;
	    checkSum.z += hForces[i].z;
	}
	
	//for (unsigned int i = 0; i<1000; i++)
	//{
		reduce<<<ksReduce.dimGrid, ksReduce.dimBlock>>>(dForces, NO_FORCES, dFinalForce);
		//add<<<ksAdd.dimGrid, ksAdd.dimBlock>>>(dFinalForce, NO_RAIN_DROPS, dDrops);
	//}

    checkDeviceMatrix<float>((float*)dFinalForce, sizeof(float3), 1, 3, "%5.2f ", "Final force");
	checkHostMatrix<float>((float*)&checkSum, sizeof(float3), 1, 3, "%5.2f ", "Check force");


	// checkDeviceMatrix<float>((float*)dDrops, sizeof(float3), NO_RAIN_DROPS, 3, "%5.2f ", "Final Rain Drops");

	if (hForces)
		free(hForces);
	if (hDrops)
		free(hDrops);

	hipFree(dForces);
	hipFree(dDrops);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);

	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	printf("Time to get device properties: %f ms", elapsedTime);
}
