#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <hipblas.h>

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

hipblasStatus_t status = hipblasStatus_t();
hipblasHandle_t handle = hipblasHandle_t();

const unsigned int N = 5;
const unsigned int dim = 3;
const unsigned int MEMSIZE = N * dim * sizeof(float);
const unsigned int THREAD_PER_BLOCK = 128;
const unsigned int GRID_SIZE = (N * dim + THREAD_PER_BLOCK - 1)/THREAD_PER_BLOCK;

void fillData(float *data, const unsigned int length, const unsigned int dim) {
	unsigned int id = 0;
	for (unsigned int i=0; i<length; i++) {
		for (unsigned int j=0; j<dim; j++) {
			data[id++]= i & 255;   // =i%256
		}
	}
}

void fillDataWithNumber(float *data, const unsigned int length, const unsigned int dim, const float number) {
	unsigned int id = 0;
	for (unsigned int i=0; i<length; i++) {
		for (unsigned int j=0; j<dim; j++) {
			data[id++]= number;
		}
	}
}

__global__ void kernelPowerTwo(const float * __restrict__ a, const float * __restrict__ b, const unsigned int length, float * __restrict__ a2, float * __restrict__ b2) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int jump = gridDim.x * blockDim.x;

    while (idx < length) {
        float tmp = b[idx];
        a2[idx] = tmp * tmp;
        tmp = b[idx];
        b2[idx] = tmp * tmp;;
        idx += jump;
    }

}

int main(int argc, char *argv[]) {
	initializeCUDA(deviceProp);
	status = hipblasCreate(&handle) ;
		
	float alpha, beta;
	float *a, *b, *m;
	float *da, *da2, *db, *db2, *dm;
	float *ones, *dones;

	// paged-locked allocation
	hipHostAlloc((void**)&a, MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&b, MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&ones, MEMSIZE,hipHostMallocDefault);
	hipHostAlloc((void**)&m, N * N * sizeof(float),hipHostMallocDefault);

	hipMalloc( (void**)&da, MEMSIZE );
	hipMalloc( (void**)&da2, MEMSIZE );
	hipMalloc( (void**)&db, MEMSIZE );
	hipMalloc( (void**)&db2, MEMSIZE );
	hipMalloc( (void**)&dones, MEMSIZE );
	hipMalloc( (void**)&dm, N * N * sizeof(float));

	fillData(a, N, dim);
	fillData(b, N, dim);
	fillDataWithNumber(ones, N, dim, 1.0f);
	
	//Copy data to DEVICE
	hipMemcpy(da, a, MEMSIZE, hipMemcpyHostToDevice);
	hipMemcpy(db, b, MEMSIZE, hipMemcpyHostToDevice);
	hipMemcpy(dones, ones, MEMSIZE, hipMemcpyHostToDevice);

	//Process a -> a^2  and b->b^2
	kernelPowerTwo<<<GRID_SIZE, THREAD_PER_BLOCK>>>(da, db, N * dim, da2, db2);

	// TODO 2: Process a^2 + b^2 using CUBLAS //pair-wise operation such that the result is dm[N*N] matrix
	// T -> transponuj
	// N -> nedělej nic
	//N,N,dim -> rozměry těch matic
	alpha = 1.0f;
	beta = 0.0f;
	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, dim, &alpha, da2, dim, dones, dim, &beta, dm, N);
    checkDeviceMatrix<float>(dm,	sizeof(float)*N, N, N, "%f ", "M");

    //alpha = 1.0f;
    beta = 1.0f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, dim, &alpha, dones, dim, db2, dim, &beta, dm, N);
	
	//TODO 3: Process -2ab and sum with previous result stored in dm using CUBLAS
    alpha = -2.0f;
    //beta = 1.0f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, N, dim, &alpha, da, dim, db, dim, &beta, dm, N);

	checkDeviceMatrix<float>(da,	sizeof(float)*dim, N, dim, "%f ", "A");
	checkDeviceMatrix<float>(da2,	sizeof(float)*dim, N, dim, "%f ", "A^2");
	checkDeviceMatrix<float>(db,	sizeof(float)*dim, N, dim, "%f ", "B");
	checkDeviceMatrix<float>(db2,   sizeof(float)*dim, N, dim, "%f ", "B^2");
	checkDeviceMatrix<float>(dones, sizeof(float)*dim, N, dim, "%f ", "ONES");
	checkDeviceMatrix<float>(dm,	sizeof(float)*N, N, N, "%f ", "M");

	
	hipFree(da);
	hipFree(da2);
	hipFree(db);
	hipFree(db2);
	hipFree(dm);
	hipFree(dones);
	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(m);
	hipHostFree(ones);

	status = hipblasDestroy(handle);
}
